#include "hip/hip_runtime.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include "utils.h"
#include <stdio.h>

constexpr float RADIUS = 60;
constexpr float SIGMA = 4;
constexpr float DBL_SIGMA_SQ = (2 * SIGMA * SIGMA);
constexpr float MIN_DISTANCE = 60;
constexpr size_t NUM_ITER = 50;
constexpr float DIST_TO_REAL = 10;
// Dataset
const std::string PATH_TO_DATA = "../../../datas/1000/random_pts_1k.csv";
const std::string PATH_TO_CENTROIDS = "../../datas/1000/random_cts_1k.csv";
constexpr int N = 5000;
constexpr int D = 3;
constexpr int M = 3;
// Device
constexpr int THREADS = 64;
constexpr int BLOCKS = (N + THREADS - 1) / THREADS;
constexpr int TILE_WIDTH = THREADS;

__global__ void mean_shift_tiling(const float* data, float* data_next) {

    // Shared memory allocation
    __shared__ float local_data[TILE_WIDTH * D];
    __shared__ float valid_data[TILE_WIDTH];
    // A few convenient variables
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int row = tid * D;
    int local_row = threadIdx.x * D;
    float new_position[D] = {0.};
    float tot_weight = 0.;
    // Load data in shared memory
    for (int t = 0; t < BLOCKS; ++t) {
        int tid_in_tile = t * TILE_WIDTH + threadIdx.x;
        if (tid_in_tile < N) {
            int row_in_tile = tid_in_tile * D;
            for (int j = 0; j < D; ++j) {
                local_data[local_row + j] = data[row_in_tile + j];
            }
            valid_data[threadIdx.x] = 1;
        }
        else {
            for (int j = 0; j < D; ++j) {
                local_data[local_row + j] = 0;
                valid_data[threadIdx.x] = 0;
            }
        }
        __syncthreads();
        for (int i = 0; i < TILE_WIDTH; ++i) {
            int local_row_tile = i * D;
            float valid_radius = RADIUS * valid_data[i];
            float sq_dist = 0.;
            for (int j = 0; j < D; ++j) {
                sq_dist += (data[row + j] - local_data[local_row_tile + j]) * (data[row + j] - local_data[local_row_tile + j]);
            }
            if (sq_dist <= valid_radius) {
                float weight = expf(-sq_dist / DBL_SIGMA_SQ);
                for (int j = 0; j < D; ++j) {
                    new_position[j] += (weight * local_data[local_row_tile + j]);
                }
                tot_weight += (weight * valid_data[i]);
            }
        }
        __syncthreads();
    }
    if (tid < N) {
        for (int j = 0; j < D; ++j) {
            data_next[row + j] = new_position[j] / tot_weight;
        }
    }
    return;
}

int main() {
    utils_ns::print_info(PATH_TO_DATA, N, D, BLOCKS, THREADS, TILE_WIDTH);
    std::cout << "Loading csv" << std::endl;
    // Load data
    std::array<float, N * D> data = utils_ns::load_csv<N, D>(PATH_TO_DATA, ',');
    std::array<float, N * D> data_next {};
    std::cout << "Csv loaded" << std::endl;
    float *dev_data;
    float *dev_data_next;
    // Allocate GPU memory
    size_t data_bytes = N * D * sizeof(float);
    hipMalloc(&dev_data, data_bytes);
    hipMalloc(&dev_data_next, data_bytes);
    // Copy to GPU memory
    hipMemcpy(dev_data, data.data(), data_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_data_next, data_next.data(), data_bytes, hipMemcpyHostToDevice);
    // Run mean shift clustering and time the execution
    std::cout << "Ended memcopy. starting ms" << std::endl;
    const auto before = std::chrono::system_clock::now();
    for (size_t i = 0; i < NUM_ITER; ++i) {
        mean_shift_tiling<<<BLOCKS, THREADS>>>(dev_data, dev_data_next);
        hipDeviceSynchronize();
        utils_ns::swap(dev_data, dev_data_next);
    }
    hipMemcpy(data.data(), dev_data, data_bytes, hipMemcpyDeviceToHost);
    const auto centroids = utils_ns::reduce_to_centroids<N, D>(data, MIN_DISTANCE);
    const auto after = std::chrono::system_clock::now();
    const std::chrono::duration<double, std::milli> duration = after - before;
    std::cout << "\nShared Memory took " << duration.count() << " ms\n" << std::endl;
    // Copy from GPU and de-allocate
    hipFree(dev_data);
    hipFree(dev_data_next);
    utils_ns::print_data<D>(centroids);
    // Check if correct number
    assert(centroids.size() == M);
    // Check if these centroids are sufficiently close to real ones
    const std::array<float, M * D> real = utils_ns::load_csv<M, D>(PATH_TO_CENTROIDS, ',');
    const bool are_close = utils_ns::are_close_to_real<M, D>(centroids, real, DIST_TO_REAL);
    assert(are_close);
    std::cout << "SUCCESS!\n";

    return 0;
}