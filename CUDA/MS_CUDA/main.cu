#include "hip/hip_runtime.h"
#include <chrono>
#include <hip/hip_runtime.h>
#include <iostream>
#include "utils.h"
#include <stdio.h>
#include <iomanip>

// Printing params
const int CONSOLE_WIDTH = 57;

// Mean shift params
const float RADIUS = 60;
const float SIGMA = 4;
const float SIGMA_POWER = (2 * SIGMA * SIGMA);
const float MIN_DISTANCE = 60;
const size_t NUM_ITER = 50;
const float EPSILON_CHECK_CENTROIDS = 10;

// Dataset
const int D = 2;
const int CENTROIDS_NUMBER = 3;
const int POINTS_NUMBER = 10000;

// Device
const int THREADS = 512;
const int TILE_WIDTH = THREADS;

__global__ void mean_shift_naive(float *data, float *data_tmp, const int POINTS_NUMBER) {
    size_t tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    if (tid < POINTS_NUMBER) {
        size_t row = tid * D;
        float new_position[D] = {0.};
        float tot_weight = 0.;
        for (size_t i = 0; i < POINTS_NUMBER; ++i) {
            size_t row_n = i * D;
            float sq_dist = 0.;
            for (size_t j = 0; j < D; ++j) {
                sq_dist += (data[row + j] - data[row_n + j]) * (data[row + j] - data[row_n + j]);
            }
            if (sq_dist <= RADIUS) {
                float weight = expf(-sq_dist / SIGMA_POWER);
                for (size_t j = 0; j < D; ++j) {
                    new_position[j] += weight * data[row_n + j];
                }
                tot_weight += weight;
            }
        }
        for (size_t j = 0; j < D; ++j) {
            data_tmp[row + j] = new_position[j] / tot_weight;
        }
    }
    return;
}

__global__ void mean_shift_tiling(const float* data, float* data_next, const int POINTS_NUMBER, const int BLOCKS) {

    // Shared memory allocation
    __shared__ float local_data[TILE_WIDTH * D];
    __shared__ float valid_data[TILE_WIDTH];
    // A few convenient variables
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;
    int row = tid * D;
    int local_row = threadIdx.x * D;
    float new_position[D] = {0.};
    float tot_weight = 0.;
    // Load data in shared memory
    for (int t = 0; t < BLOCKS; ++t) {
        int tid_in_tile = t * TILE_WIDTH + threadIdx.x;
        if (tid_in_tile < POINTS_NUMBER) {
            int row_in_tile = tid_in_tile * D;
            for (int j = 0; j < D; ++j) {
                local_data[local_row + j] = data[row_in_tile + j];
            }
            valid_data[threadIdx.x] = 1;
        }
        else {
            for (int j = 0; j < D; ++j) {
                local_data[local_row + j] = 0;
                valid_data[threadIdx.x] = 0;
            }
        }
        __syncthreads();
        for (int i = 0; i < TILE_WIDTH; ++i) {
            int local_row_tile = i * D;
            float valid_radius = RADIUS * valid_data[i];
            float sq_dist = 0.;
            for (int j = 0; j < D; ++j) {
                sq_dist += (data[row + j] - local_data[local_row_tile + j]) * (data[row + j] - local_data[local_row_tile + j]);
            }
            if (sq_dist <= valid_radius) {
                float weight = expf(-sq_dist / SIGMA_POWER);
                for (int j = 0; j < D; ++j) {
                    new_position[j] += (weight * local_data[local_row_tile + j]);
                }
                tot_weight += (weight * valid_data[i]);
            }
        }
        __syncthreads();
    }
    if (tid < POINTS_NUMBER) {
        for (int j = 0; j < D; ++j) {
            data_next[row + j] = new_position[j] / tot_weight;
        }
    }
    return;
}

std::string separation_line(){
    return std::string(CONSOLE_WIDTH, '-');
}

std::string console_log(std::string log){
    int spaces = CONSOLE_WIDTH-log.length()-2;
    return "|" + log + std::string(spaces>0?spaces:0, ' ') + "|";
}

std::string console_log_time(std::string log, const std::chrono::duration<double, std::milli> duration){
    return console_log(log + std::to_string(duration.count()) + "ms");
}

int execute_mean_shift(bool USE_SHARED) {
    const int BLOCKS = (POINTS_NUMBER + THREADS - 1) / THREADS;

    // Print useful infos
    std::cout << separation_line() << std::endl;
    std::cout << console_log(USE_SHARED?"CUDA MEAN SHIFT: SHARED MEMORY":"CUDA MEAN SHIFT: NAIVE") << std::endl;
    std::cout << separation_line() << std::endl;
    std::cout << "|POINTS_NUMBER\t|BLOCKS\t|THREADS\t|TILE_WIDTH\t|"<<std::endl;
    std::cout << "|" << POINTS_NUMBER << "      \t|" << BLOCKS << "\t|" << THREADS << "      \t|" << TILE_WIDTH << "      \t!"<<std::endl;
    std::cout << separation_line() << std::endl;

    //Compute paths
    const std::string PATH_TO_DATA = "../../datas/"+std::to_string(POINTS_NUMBER)+"/points.csv";
    const std::string PATH_TO_CENTROIDS = "../../datas/"+std::to_string(POINTS_NUMBER)+"/centroids.csv";

    const auto start_prog = std::chrono::system_clock::now();

    // Load data
    std::cout << console_log("Loading csv...") << std::endl;
    std::array<float, POINTS_NUMBER * D> data = utils_ns::load_csv<POINTS_NUMBER, D>(PATH_TO_DATA, ',');
    std::array<float, POINTS_NUMBER * D> data_next {};
    std::cout << console_log("Done") << std::endl;
    std::cout << separation_line() << std::endl;

    // Allocate GPU memory
    float *dev_data, *dev_data_tmp;
    size_t data_bytes = POINTS_NUMBER * D * sizeof(float);
    hipMalloc(&dev_data, data_bytes);
    hipMalloc(&dev_data_tmp, data_bytes);

    // Copy to GPU memory
    const auto start_memcp = std::chrono::system_clock::now();
    hipMemcpy(dev_data, data.data(), data_bytes, hipMemcpyHostToDevice);
    hipMemcpy(dev_data_tmp, data_next.data(), data_bytes, hipMemcpyHostToDevice);
    const std::chrono::duration<double, std::milli> duration_memcp = std::chrono::system_clock::now() - start_memcp;
    std::cout << console_log_time("Ended memcopy in ", duration_memcp) << std::endl;


    // Run mean shift clustering and time the execution
    std::cout << separation_line() << std::endl;
    std::cout << console_log("Executing mean shift...") << std::endl;

    const auto starting_mean_shift_time = std::chrono::system_clock::now();
    for (size_t i = 0; i < NUM_ITER; ++i) {
        if(USE_SHARED){
            mean_shift_tiling<<<BLOCKS, THREADS>>>(dev_data, dev_data_tmp, POINTS_NUMBER, BLOCKS);
        }else{
            mean_shift_naive<<<BLOCKS, THREADS>>>(dev_data, dev_data_tmp, POINTS_NUMBER);
        }
        hipDeviceSynchronize();
        utils_ns::swap(dev_data, dev_data_tmp);
    }
    hipMemcpy(data.data(), dev_data, data_bytes, hipMemcpyDeviceToHost);
    const auto centroids = utils_ns::reduce_to_centroids<POINTS_NUMBER, D>(data, MIN_DISTANCE);
    const std::chrono::duration<double, std::milli> duration_mean_shift = std::chrono::system_clock::now() - starting_mean_shift_time;
    std::cout << console_log_time("Duration: ", duration_mean_shift) << std::endl;

    // Copy from GPU and de-allocate
    hipFree(dev_data);
    hipFree(dev_data_tmp);
    std::cout << separation_line() << std::endl;
    std::cout << console_log("Centroids found:") << std::endl;
    for (const auto& c : centroids) {
        std::string xy = std::to_string(c[0]) + ", " + std::to_string(c[1]);
        std::cout << console_log(xy) << std::endl;
    }
    std::cout << separation_line() << std::endl;

    // Check if correct number
    if (centroids.size() != CENTROIDS_NUMBER){
        std::cout << console_log("ERROR: resulting centroids number are different from originals!") << std::endl;
        return 1;
    }

    // Check if these centroids are sufficiently close to real ones
    const std::array<float, CENTROIDS_NUMBER * D> real = utils_ns::load_csv<CENTROIDS_NUMBER, D>(PATH_TO_CENTROIDS, ',');
    const bool are_close = utils_ns::are_close_to_real<CENTROIDS_NUMBER, D>(centroids, real, EPSILON_CHECK_CENTROIDS);
    if (!utils_ns::are_close_to_real<CENTROIDS_NUMBER, D>(centroids, real, EPSILON_CHECK_CENTROIDS)){
        std::cout << console_log("ERROR: resulting centroids are too different from originals!") << std::endl;
        return 2;
    }

    // Show execution time
    const std::chrono::duration<double, std::milli> duration_all = std::chrono::system_clock::now() - start_prog;
    std::cout << console_log_time("PROCESS ENDED in ", duration_all) << std::endl;
    std::cout << separation_line() << std::endl;
    return 0;
}


int main(int argc, char *argv[]){
    const int res1 = execute_mean_shift(false);
    std::cout << std::endl;
    std::cout << std::endl;
    const int res2 = execute_mean_shift(true);
    return res1 + res2;
}